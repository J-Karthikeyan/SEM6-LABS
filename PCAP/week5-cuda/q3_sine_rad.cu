#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

__global__ void sine_of_rad(float *rad, float *sine, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n){
        sine[tid] = sin(rad[tid]); 
    }
}

int main(void){
    float *rad, *sine;
    float *d_rad, *d_sine;
    int N = 2048;
    size_t size = N * sizeof(float);

    rad = (float *)malloc(size);
    sine = (float *)malloc(size);
    
    hipMalloc((void **)&d_rad, size);
    hipMalloc((void **)&d_sine, size);

    for(int i=0; i<N; i++){
        rad[i] = 0.001 * i;
    }
    
    int threads_per_block = 256;
    // equivalent to ceil[N / 256]
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    hipMemcpy(d_rad, rad, size, hipMemcpyHostToDevice);
    
    sine_of_rad<<<blocks_per_grid, threads_per_block>>>(d_rad, d_sine, N);
    
    hipMemcpy(sine, d_sine, size, hipMemcpyDeviceToHost);
    
    // sample 10 results 
    for(int i=1; i<N; i+=128){
        printf("sine[%f] = %f\n", rad[i], sine[i]);
    }

    hipFree(d_rad);
    hipFree(d_sine);
    free(rad);
    free(sine);

    return 0;
}
