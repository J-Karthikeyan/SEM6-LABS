#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void vector_add(int* A, int* B, int* C, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) {
        C[tid] = A[tid] + B[tid];
    }
}

int main(void){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    const int N = 10;
    size_t size = N * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    int flag = 0;
    printf("0 - (a) N blocks | 1 - (b) N threads : ");
    scanf("%d", &flag);
    printf("Enter %d integers for vector a: ", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &a[i]);
    }

    printf("Enter %d integers for vector b: ", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &b[i]);
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    if (flag == 0) {
        vector_add<<<N, 1>>>(d_a, d_b, d_c, N);
    } else {
        vector_add<<<1, N>>>(d_a, d_b, d_c, N);
    }

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("c[%d] = %d\n", i, c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
