#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA kernel definition
__global__ void vector_add(float* A, float* B, float* C, int n)
{
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we don't exceed array bounds
    if(tid < n) {
        C[tid] = A[tid] + B[tid];
    }
}

int main(void)
{
    const int N = 1 << 20;  // 1 million elements (1048576)
    size_t size = N * sizeof(float);
    float *h_A, *h_B, *h_C;  // Host pointers
    float *d_A, *d_B, *d_C;  // Device pointers

    // 1. Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // 2. Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // 3. Initialize host arrays
    for(int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // 4. Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 5. Configure and launch kernel
    int threads_per_block = 256;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    
    vector_add<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_C, N);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();

    // 6. Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 7. Verify result
    for(int i = 0; i < 10; i++) {
        printf("h_C[%d] = %.2f\n", i, h_C[i]);
    }

    // 8. Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
