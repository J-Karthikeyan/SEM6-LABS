#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(int *A, int *B, int *C, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) 
        C[tid] = A[tid] + B[tid];
    
}

int main(void){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int N = 2048;
    size_t size = N * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = 2 * i + 1;
    }

    int threads_per_block = 256;
    // equivalent to ceil[N / 256]
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    vector_add<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, N);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    // sample 10 results 
    for(int i=0; i<10; i++){
        printf("c[%d] = %d\n", i, c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

}
